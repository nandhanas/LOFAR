#include "hip/hip_runtime.h"
#ifdef _OPENMP
#include <omp.h>
#endif
#include "w-stacking.h"
#include <math.h>
#include <stdlib.h>
#include <stdio.h>

#ifdef __HIPCC__

__global__ void phase_g(int xaxis, 
		        int yaxis,
			int num_w_planes,
			double * gridss,
			double * image_real,
			double * image_imag,
			double wmin,
			double dw,
			double dwnorm,
			int xaxistot,
			int yaxistot,
			double resolution,
			int nbucket)
{
	long gid = blockIdx.x*blockDim.x + threadIdx.x;
	double add_term_real;
	double add_term_img;
	double wterm;
	long arraysize = (long)((xaxis*yaxis*num_w_planes)/nbucket);

	if(gid < arraysize)
	{
	  long gid_aux = nbucket*gid;
	  for(int iaux=0; iaux<nbucket; iaux++) 
          {
		int iw = gid_aux/(xaxis*yaxis);
		int ivaux = gid_aux%(xaxis*yaxis);
		int iv = ivaux/xaxis;
		int iu = ivaux%xaxis;
		long index = 2*gid_aux;
		long img_index = iu+iv*xaxis;

                wterm = wmin + iw*dw;

#ifdef PHASE_ON
                if (num_w_planes > 1)
                {
                    double xcoord = (double)(iu-xaxistot/2);
                    if(xcoord < 0.0)xcoord = (double)(iu+xaxistot/2);
                    xcoord = sin(xcoord*resolution);
                    double ycoord = (double)(iv-yaxistot/2);
                    if(ycoord < 0.0)ycoord = (double)(iv+yaxistot/2);
                    ycoord = sin(ycoord*resolution);

                    double preal, pimag;
                    double radius2 = (xcoord*xcoord+ycoord*ycoord);

                    preal = cos(2.0*PI*wterm*(sqrt(1-radius2)-1.0));
                    pimag = sin(2.0*PI*wterm*(sqrt(1-radius2)-1.0));

                    double p,q,r,s;
                    p = gridss[index];
                    q = gridss[index+1];
                    r = preal;
                    s = pimag;

                    //printf("%d %d %d %ld %ld\n",iu,iv,iw,index,img_index);

		    add_term_real = (p*r-q*s)*dwnorm*sqrt(1-radius2);
		    add_term_img = (p*s+q*r)*dwnorm*sqrt(1-radius2);
		    atomicAdd(&(image_real[img_index]),add_term_real);
		    atomicAdd(&(image_imag[img_index]),add_term_img);
                } else {
		    atomicAdd(&(image_real[img_index]),gridss[index]);
		    atomicAdd(&(image_imag[img_index]),gridss[index+1]);
                }
#else
		atomicAdd(&(image_real[img_index]),gridss[index]);
		atomicAdd(&(image_imag[img_index]),gridss[index+1]);
#endif // end of PHASE_ON
		gid_aux++;
           }
	}

}

#endif

void phase_correction(double* gridss, double* image_real, double* image_imag, int xaxis, int yaxis, int num_w_planes, int xaxistot, int yaxistot,
		      double resolution, double wmin, double wmax, int num_threads)
{
        double dw = (wmax-wmin)/(double)num_w_planes;
	double wterm = wmin+0.5*dw;
	double dwnorm = dw/(wmax-wmin);

#ifdef __HIPCC__

	// WARNING: nbucket MUST be chosen such that xaxis*yaxis*num_w_planes is a multiple of nbucket
	int nbucket = 1;
        int Nth = NTHREADS;
        long Nbl = (long)((num_w_planes*xaxis*yaxis)/Nth/nbucket) + 1;
        if(NWORKERS == 1) {Nbl = 1; Nth = 1;};
        printf("Running on GPU with %d threads and %d blocks\n",Nth,Nbl);
	

	hipError_t mmm;
	double * image_real_g;
	double * image_imag_g;
	double * gridss_g;

        mmm=hipMalloc(&gridss_g, 2*num_w_planes*xaxis*yaxis*sizeof(double));
	//printf("CUDA ERROR 1 %s\n",hipGetErrorString(mmm));
	mmm=hipMalloc(&image_real_g, xaxis*yaxis*sizeof(double));
	//printf("CUDA ERROR 2 %s\n",hipGetErrorString(mmm));
	mmm=hipMalloc(&image_imag_g, xaxis*yaxis*sizeof(double));
	//printf("CUDA ERROR 3 %s\n",hipGetErrorString(mmm));

	mmm=hipMemcpy(gridss_g, gridss, 2*num_w_planes*xaxis*yaxis*sizeof(double), hipMemcpyHostToDevice);
	//printf("CUDA ERROR 4 %s\n",hipGetErrorString(mmm));
	mmm=hipMemset(image_real_g, 0.0, xaxis*yaxis*sizeof(double));
	//printf("CUDA ERROR 5 %s\n",hipGetErrorString(mmm));
	mmm=hipMemset(image_imag_g, 0.0, xaxis*yaxis*sizeof(double));
	//printf("CUDA ERROR 6 %s\n",hipGetErrorString(mmm));

	// call the phase correction kernel
	phase_g <<<Nbl,Nth>>> (xaxis,
                               yaxis,
			       num_w_planes,
                               gridss_g,
                               image_real_g,
                               image_imag_g,
                               wmin,
                               dw,
                               dwnorm,
                               xaxistot,
                               yaxistot,
                               resolution,
			       nbucket);

	mmm = hipMemcpy(image_real, image_real_g, xaxis*yaxis*sizeof(double), hipMemcpyDeviceToHost);
	//printf("CUDA ERROR 7 %s\n",hipGetErrorString(mmm));
	mmm = hipMemcpy(image_imag, image_imag_g, xaxis*yaxis*sizeof(double), hipMemcpyDeviceToHost);
	//printf("CUDA ERROR 8 %s\n",hipGetErrorString(mmm));

#else

#ifdef _OPENMP
    omp_set_num_threads(num_threads);
#endif

        #pragma omp parallel for collapse(3) private(wterm)
	for (int iw=0; iw<num_w_planes; iw++)
	{
	    for (int iv=0; iv<yaxis; iv++)
            for (int iu=0; iu<xaxis; iu++)
            {

		long index = 2*(iu+iv*xaxis+xaxis*yaxis*iw);
		long img_index = iu+iv*xaxis;
		wterm = wmin + iw*dw;
#ifdef PHASE_ON
                if (num_w_planes > 1)
		{
                    double xcoord = (double)(iu-xaxistot/2);
                    if(xcoord < 0.0)xcoord = (double)(iu+xaxistot/2);
		    xcoord = sin(xcoord*resolution);
                    double ycoord = (double)(iv-yaxistot/2);
                    if(ycoord < 0.0)ycoord = (double)(iv+yaxistot/2);
		    ycoord = sin(ycoord*resolution);

		    double preal, pimag;
		    double radius2 = (xcoord*xcoord+ycoord*ycoord);
		    if(xcoord <= 1.0)
		    {
			    preal = cos(2.0*PI*wterm*(sqrt(1-radius2)-1.0));
			    pimag = sin(2.0*PI*wterm*(sqrt(1-radius2)-1.0));
		    } else {
			    preal = cos(-2.0*PI*wterm*(sqrt(radius2-1.0)-1));
			    pimag = 0.0;
		    }

		    preal = cos(2.0*PI*wterm*(sqrt(1-radius2)-1.0));
		    pimag = sin(2.0*PI*wterm*(sqrt(1-radius2)-1.0));

		    double p,q,r,s;
		    p = gridss[index];
		    q = gridss[index+1];
		    r = preal;
		    s = pimag;

		    //printf("%d %d %d %ld %ld\n",iu,iv,iw,index,img_index);
		    #pragma omp atomic
		    image_real[img_index] += (p*r-q*s)*dwnorm*sqrt(1-radius2);
		    #pragma omp atomic
		    image_imag[img_index] += (p*s+q*r)*dwnorm*sqrt(1-radius2);
	        } else {
		    #pragma omp atomic
		    image_real[img_index] += gridss[index];
		    #pragma omp atomic
		    image_imag[img_index] += gridss[index+1];
		}
#else
		#pragma omp atomic
  	        image_real[img_index] += gridss[index];
		#pragma omp atomic
		image_imag[img_index] += gridss[index+1];
#endif // end of PHASE_ON

            }
	}

#endif // end of __HIPCC__


}
